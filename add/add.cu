#include "hip/hip_runtime.h"
#include <iostream.cpp>
#include <math.h>

//function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{   
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index, i < n, i+=stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = i << 20; //1 M elements
    float *x,float *y ;
    //Allocate Unified Memory - accessible from CPU or GPU
    hipMallocManaged(&x,N*sizeof(float));
    hipMallocManaged(&y,N*sizeof(float));
    
    //intialize x and y arrays on the host
    for (int i = 0; i < n; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    //Run kernel on 1M elements on the CPU
    add<<<1, 256>>>(N, x, y);

    //Wait for GPU to finish before accesing on host
    hipDeviceSynchronize();

    //Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
